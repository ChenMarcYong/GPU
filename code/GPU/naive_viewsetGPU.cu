#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <iomanip>

#include "naive_viewsetGPU.hpp"



using namespace std;
#define ThrPerBlock_y 8
#define ThrPerBlock_x 8

__device__ float calculateAngleNaiveGPU(float Dz, float Dx, float Dy)
{
    float dist = sqrt( (Dx) * (Dx) + (Dy) * (Dy));
    return atan(Dz / dist);
}

__device__ uint8_t DDANaiveGPU(const uint8_t *dev_in, int Px, int Py, const int Cx, const int Cy, const int MapWidth)
{
    int Dx, Dy, D;  // delta
    Dx = Px - Cx;
    Dy = Py - Cy;

    int Dz = dev_in[Py * MapWidth + Px] - dev_in[Cy * MapWidth + Cx];
    D = max(abs(Dx), abs(Dy));
    float angleRef;
    angleRef = calculateAngleNaiveGPU(Dz, Dx, Dy);
    //angleRef = atan(Dz / __fsqrt_rn( (Dx * Dx) + (Dy * Dy)  )); 
    float angleDDA;

    float stepX, stepY;
    stepX = (float(Dx) / D);
    stepY = (float(Dy) / D);

    int DDAx, DDAy;
    for(int i = 0; i < D; i++)
    {
        DDAx = Cx + i * stepX; 
        DDAy = Cy + i * stepY;
        Dx = Px - DDAx;
        Dy = Py - DDAy;
        Dz = dev_in[Py * MapWidth + Px] - dev_in[DDAy * MapWidth + DDAx];
        //angleDDA = calculateAngleNaiveGPU(Dz, Dx, Dy);
        angleDDA = atan(Dz / __fsqrt_rn( (Dx * Dx) + (Dy * Dy)  )); 

        if (angleRef > angleDDA) return 0;
    }
    return 255;
}

__global__ void kernelNaive_viewsetGPU(const uint8_t *dev_in, uint8_t *dev_out, int Cx, int Cy, const int MapHeight, const int MapWidth)
{

    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    int initX = indexX;

    while (indexY < MapHeight)
    {
        while (indexX < MapWidth)
        {
            dev_out[indexY * MapWidth + indexX] = DDANaiveGPU(dev_in, indexX, indexY, Cx, Cy, MapWidth);
            indexX += gridDim.x * blockDim.x;
        }

        indexY += gridDim.y * blockDim.y;
        indexX = initX;
    }
}

void naive_viewsetGPU(const uint8_t *h_in, uint8_t *h_out, int Cx, int Cy, const int MapHeight, const int MapWidth)
{
    uint8_t *dev_in, *dev_out;

    HANDLE_ERROR(hipMalloc(&dev_in, sizeof(uint8_t) * MapHeight * MapWidth));
    HANDLE_ERROR(hipMalloc(&dev_out, sizeof(uint8_t) * MapHeight * MapWidth));

    HANDLE_ERROR(hipMemcpy(dev_in, h_in, sizeof(uint8_t) * MapHeight * MapWidth, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_out, h_out, sizeof(uint8_t) * MapHeight * MapWidth, hipMemcpyHostToDevice));


    int blocks_x = (MapWidth + ThrPerBlock_x - 1) / ThrPerBlock_x;
    int blocks_y = (MapHeight + ThrPerBlock_y - 1) / ThrPerBlock_y;

    dim3 gridDim(blocks_x, blocks_y);
    dim3 blockDim(ThrPerBlock_x, ThrPerBlock_y);

    kernelNaive_viewsetGPU<<<gridDim, blockDim >>>(dev_in, dev_out, Cx, Cy, MapHeight, MapWidth);

    HANDLE_ERROR(hipMemcpy(h_out, dev_out, sizeof(uint8_t) * MapHeight * MapWidth, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(dev_in));
    HANDLE_ERROR(hipFree(dev_out));
}


