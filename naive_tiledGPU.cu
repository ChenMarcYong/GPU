#include "hip/hip_runtime.h"
#include "utils/commonCUDA.hpp"
#include "utils/chronoGPU.hpp"

#include "naive_tiledGPU.hpp"
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <iomanip>

#include "utils/chronoCPU.hpp"
#include "utils/chronoGPU.hpp"

using namespace std;
#define ThrPerBlock_y 8
#define ThrPerBlock_x 8

__global__ void kernelTiled(uint8_t *h_in, uint8_t *h_out,const int inMapWidth, const int inMapHeight , const int outMapWidth, const int outMapHeight)
{
    int TiledDimX = int(inMapWidth / outMapWidth);
    int TiledDimY = int(inMapHeight / outMapHeight);

    __shared__ int max;

    int location = (blockIdx.y * TiledDimY) * (inMapWidth) + blockIdx.x * TiledDimX;

    for(int Py =threadIdx.y; Py < TiledDimY; Py += blockDim.y)
    {
        for(int Px =threadIdx.x; Px < TiledDimX; Px += blockDim.x)
        {
            int point = location + Py * inMapWidth + Px; 
            atomicMax(&max, int(h_in[point]));
        }               
    }
    __syncthreads();
    h_out[blockIdx.y * outMapWidth + blockIdx.x] = max;
}


void TiledGPU(uint8_t *h_in, uint8_t *h_out,const int inMapWidth, const int inMapHeight , const int TileDimX, const int TileDimY)
{
    ChronoGPU chrGPU;
    uint8_t *dev_h_in;
    uint8_t *dev_h_out;

    size_t sizeIn = inMapWidth * inMapHeight * sizeof(uint8_t);
    size_t sizeOut = TileDimX * TileDimY * sizeof(uint8_t);

    hipMalloc((void**) &dev_h_in, sizeIn);
    hipMalloc((void**) &dev_h_out, sizeOut);

    hipMemcpy(dev_h_in, h_in, sizeIn, hipMemcpyHostToDevice);
    


    dim3 gridDim(TileDimX, TileDimY);
    dim3 blockDim(ThrPerBlock_x, ThrPerBlock_y);
    ChronoGPU chrk1;
    chrk1.start();
    kernelTiled<<<gridDim, blockDim>>>(dev_h_in, dev_h_out, inMapWidth, inMapHeight, TileDimX, TileDimY);
    chrk1.stop();

	const float timeComputechrk1 = chrk1.elapsedTime();
	//std::cout << "-> Done kernelAngle : " << chrk1 << " ms" << std::endl;
    printf("Done kernelTiled : %f ms\n", timeComputechrk1);


    hipMemcpy(h_out, dev_h_out, sizeOut, hipMemcpyDeviceToHost);

    hipFree(dev_h_in);
    hipFree(dev_h_out);
}